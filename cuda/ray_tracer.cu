
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 		16
#define MAX_TRIANGLE	100
#define MAX_LIGHT 		10
#define eps 			0.0001
#define MAX_RAY_DEPTH 	20
#define	AAKERNEL_SIZE	6


using namespace std;

__device__ unsigned int WangHash(unsigned int a) {
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Vec {        
   float x, y, z;                 
   __host__ __device__ Vec(){} 
   __host__ __device__ Vec(float x_, float y_, float z_){ x=x_; y=y_; z=z_; } 
   __host__ __device__ Vec operator+(const Vec &b) const { return Vec(x+b.x,y+b.y,z+b.z); } 
   __host__ __device__ Vec operator-(const Vec &b) const { return Vec(x-b.x,y-b.y,z-b.z); } 
   __host__ __device__ Vec operator*(float b) const { return Vec(x*b,y*b,z*b); } 
   __host__ __device__ Vec mult(const Vec &b) const { return Vec(x*b.x,y*b.y,z*b.z); } 
   __host__ __device__ Vec& norm(){ return *this = *this * (1/sqrtf(x*x+y*y+z*z)); } 
   __host__ __device__ float dot(const Vec &b) const { return x*b.x+y*b.y+z*b.z; } // cross: 
   __host__ __device__ Vec operator%(const Vec&b) const{return Vec(y*b.z-z*b.y,z*b.x-x*b.z,x*b.y-y*b.x);} 
}; 

__device__ float normalize(Vec v)
{
	return sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
}

struct Parameter {
	int w;
	int h;
	int samps;
	int n_triangles;
	int n_lights;
	float fov;
    float aspectratio;
	float angle;
};

struct Tracing_Stack{
	Vec o;
	Vec d;
	Vec pre_color;
	int depth;
	__device__ Tracing_Stack(){};	
	__device__ Tracing_Stack(const Vec &o,
					  const Vec &d,
					  const Vec &pre_color,
					  const int depth){
		this->o = o;
		this->d = d;
		this->pre_color = pre_color;
		this->depth = depth;
	}
};

struct Light
{
	Vec pos;
	Vec color;
	Vec x_vec;
	Vec y_vec;
	int n_x;
	int n_y;
	
	
};


class Triangle
{
public:	
	Vec v1;   // use one vertex and two edges to save some time and space
	Vec edge1;
	Vec edge2;
	Vec norm;
	Vec surfaceColor;
	Vec emissionColor;
	char token;		// T or D or L- transparent or diffusive surface or light source surface
    __host__ __device__ Triangle(){};
	__host__ Triangle(const Vec &a,
					  const Vec &b,
					  const Vec &c, 
					  const Vec &sColor, 
					  const Vec &eColor, 
					  char tld_token){
		v1 = a;
		Vec v2 = b;
		Vec v3 = c;
		edge1 = v2-v1;
		edge2 = v3-v1;
		token = tld_token;
		surfaceColor = sColor;
		emissionColor =  eColor;
		computeNormal();
	}

	__host__ void computeNormal()
	{
		norm = edge2%edge1;
		norm.norm();		
	}

	__device__ bool intersection(const Vec &origin, const Vec &dir, float & t) const
	{
		Vec pVec = dir%edge2;
		float det=edge1.dot(pVec);
		//if(det>-eps && det <eps)
		if(det==0)
		{
			return false;
		}

		float invDet=1./det;
		Vec tVec=origin-v1;
		float u=(tVec.dot(pVec))*(invDet);
		if(u<0. || u>1.)
		{
			return false;
		}

		Vec qVec = tVec%edge1;
		float v = dir.dot(qVec)*(invDet);

		if (v<0.||v+u>1.)
		{
			return false;
		}
		t = (edge2.dot(qVec))*(invDet);
		if (t>eps){
			return true;
		}
		return false;
	}	

};

__constant__ Triangle ctriangles[MAX_TRIANGLE];
__constant__ Parameter cparam[1];
__constant__ Light clights[MAX_LIGHT];
__constant__ float AAFilter[AAKERNEL_SIZE][3] 	=		/* X, Y, coef */
	{
		-0.52, 0.38, 0.128,
		0.41, 0.56, 0.119,
		0.27, 0.08, 0.294,
		-0.17, -0.29, 0.249,
		0.58, -0.55, 0.104,
		-0.31, -0.71, 0.106
	};	
__host__ void parse(string file_name, Triangle* triangles, int &n_triangles)
{
	ifstream fin;
	fin.open(file_name);
	int cnt_v  = 0;
	float v[15];

	if(fin.fail())
	{
		cout<<"Could not open file"<<endl;
		exit(1);
	}

	string buffer;
	n_triangles = 0;
	while(!fin.eof())
	{
		getline(fin,buffer);
	    istringstream buf(buffer);
	    for(string token; getline(buf, token,' '); )
        {
        	if (token=="triangle") {
        		cnt_v = 0;
			}
			else if(token=="T" || token=="D"|| token=="S")
			{
				if (n_triangles>=MAX_TRIANGLE){
					cout<<"Number of triangles should be equal or less than "<<MAX_TRIANGLE<<endl;
					exit(1);
				}
				Vec v1(v[0],v[1],v[2]);
				Vec v2(v[3],v[4],v[5]);
				Vec v3(v[6],v[7],v[8]);
				Vec sColor(v[9],v[10],v[11]);
				Vec eColor(v[12],v[13],v[14]);
				Triangle tri_tmp(v1,v2,v3,sColor,eColor,token[0]);
				triangles[n_triangles++] = tri_tmp;
				
			}
        	else v[cnt_v++] = stof(token);
		}
	}
	fin.close();
	return;
}

__host__ float clamp(float x){ return x<0 ? 0 : x>1 ? 1 : x; }
__host__ int toInt(float x){ return int(powf(clamp(x),1/2.2)*255+.5); }


__global__ void init_rand(hiprandState *state, unsigned int seed) {
	int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = idx_y*(cparam[0].w)+idx_x;
	hiprand_init(seed + WangHash(idx),0 , 0, &state[idx]);
}


__device__ bool intersect(const Vec &origin, const Vec &direction, float &t, int &id){
	   int n = cparam[0].n_triangles;  
	   float d, inf=1e5;
	   t = inf;
	   for(int i=0; i<n; i++) if((ctriangles[i].intersection(origin,direction,d))&&d<t){t=d;id=i;} 
	   return (t<inf);
}

__device__ Vec raytrace(Vec &o, Vec &d) {
	
	Tracing_Stack stk[MAX_RAY_DEPTH+1];
	int stk_cnt = 1;
	stk[0] = Tracing_Stack(o,d,Vec(1,1,1),0);
	Vec color(0,0,0);
	
	do{
		float t;
		int id;
		if (!intersect(stk[stk_cnt-1].o, stk[stk_cnt-1].d, t, id)) {stk_cnt--; continue;} //no hit
		const Triangle &obj = ctriangles[id];        // the hit object
		Vec new_o = stk[stk_cnt-1].o + stk[stk_cnt-1].d*t;    // update the origin
		Vec n = obj.norm;
		Vec nl = n.dot(stk[stk_cnt-1].d)<0?n:n*-1;
		Vec f = obj.surfaceColor;
	   
		//float p = f.x>f.y && f.x>f.z ? f.x : f.y>f.z ? f.y : f.z; // max refl 
		if (stk[stk_cnt-1].depth>=MAX_RAY_DEPTH) {
			 {color = color + stk[stk_cnt-1].pre_color.mult(obj.emissionColor);stk_cnt--;continue;} //R.R.
		}
		
		if (obj.token == 'D'){                  // Ideal DIFFUSE reflection 
			
			Vec col(0,0,0);
	
			for (int i = 0; i < cparam[0].n_lights;i++){
				float factor = 1./clights[i].n_x/clights[i].n_y;
				for (int j = 0; j<clights[i].n_x; j++) {
					for (int k = 0; k<clights[i].n_y; k++) {
						Vec l_pos = clights[i].pos - clights[i].x_vec*0.5 + clights[i].x_vec * (1./clights[i].n_x*j) 
											  - clights[i].y_vec*0.5 + clights[i].y_vec * (1./clights[i].n_y*k);
						Vec d = (l_pos - new_o);
						float t_light = normalize(d);
						d = d.norm();
						int id = 0;
						if (!intersect(new_o, d, t, id) || ctriangles[id].token=='L' || t>t_light) {
							col = col + f.mult(clights[i].color)*(d.dot(obj.norm))*factor;	
						}
					}
				}
			}
			color = color + stk[stk_cnt-1].pre_color.mult(col);stk_cnt--;
			
			continue;
		}
		
		
		else if (obj.token == 'S'){            // Ideal SPECULAR reflection 
   			color = color+stk[stk_cnt-1].pre_color.mult(obj.emissionColor);
			stk[stk_cnt-1].o = new_o;
			stk[stk_cnt-1].d = stk[stk_cnt-1].d - n*2*n.dot(stk[stk_cnt-1].d);
			stk[stk_cnt-1].pre_color = stk[stk_cnt-1].pre_color.mult(f);
			stk[stk_cnt-1].depth++;
			continue;
		}
		
				
   		Vec reflRay(stk[stk_cnt-1].d-n*2*n.dot(stk[stk_cnt-1].d));     // Ideal dielectric REFRACTION
		
   		bool into = n.dot(nl)>0;                // Ray from outside going in? 
   		float nc=1, nt=2.4, nnt=into?nc/nt:nt/nc, ddn=stk[stk_cnt-1].d.dot(nl), cos2t; 
   		if ((cos2t=1-nnt*nnt*(1-ddn*ddn))<0)    {// Total internal reflection
     //cout<<"TIR"<<endl;
			color = color + stk[stk_cnt-1].pre_color.mult(obj.emissionColor);
			stk[stk_cnt-1].o = new_o;
			stk[stk_cnt-1].d = reflRay;
			stk[stk_cnt-1].pre_color = stk[stk_cnt-1].pre_color.mult(f);
			stk[stk_cnt-1].depth++;
			continue;
   		}
	 	Vec tdir = (stk[stk_cnt-1].d*nnt - n*((into?1:-1)*(ddn*nnt+sqrtf(cos2t)))).norm(); 
  		float a=nt-nc, b=nt+nc, R0=a*a/(b*b), c = 1-(into?-ddn:tdir.dot(n)); 
  		float Re=R0+(1-R0)*c*c*c*c*c,Tr=1-Re;
		
		color = color + stk[stk_cnt-1].pre_color.mult(obj.emissionColor);
		stk[stk_cnt-1].o = new_o;
		stk[stk_cnt-1].d = reflRay;
		stk[stk_cnt-1].pre_color = stk[stk_cnt-1].pre_color.mult(f);
		stk[stk_cnt-1].depth++;
		stk_cnt++;
		stk[stk_cnt-1].o = new_o;
		stk[stk_cnt-1].d = tdir;
		stk[stk_cnt-1].pre_color = stk[stk_cnt-2].pre_color;
		stk[stk_cnt-1].depth = stk[stk_cnt-2].depth;
		
		stk[stk_cnt-2].pre_color = stk[stk_cnt-2].pre_color*Re;
		stk[stk_cnt-1].pre_color = stk[stk_cnt-1].pre_color*Tr;
		
	} while (stk_cnt);
		
	
	
	return color;
}




__global__ void path_tracing(Vec *d_c) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = y*(cparam[0].w)+x;

	Vec dr;
	d_c[idx] = Vec(0,0,0);
	for (int i = 0; i<AAKERNEL_SIZE; i++){
		dr.x = (2. * ((x+0.5+AAFilter[i][0])/cparam[0].w) -1.  )*cparam[0].angle*cparam[0].aspectratio;
		float temp = (1. - 2.*((y+0.5+AAFilter[i][1])/cparam[0].h)) ;
		dr.y=temp*cparam[0].angle;
		dr.z = 1.;

		dr.norm();

		Vec dr_origin(0,0,0);

		d_c[idx] = d_c[idx] + (raytrace(dr_origin, dr)*AAFilter[i][2]);
	}
}

int main(int argc, char *argv[]){
	Parameter hparam;
	hparam.w = 1024;
	hparam.h = 1024;
	hparam.samps = argc==2 ? atoi(argv[1]) : 500; // # samples

	if (hparam.w%BLOCK_SIZE) {
		hparam.w = (hparam.w/BLOCK_SIZE+1)*BLOCK_SIZE;
		cout<<"Width has been changed to "<<hparam.w<<endl;
	}
	if (hparam.h%BLOCK_SIZE) {
		hparam.h = (hparam.h/BLOCK_SIZE+1)*BLOCK_SIZE;
		cout<<"Height has been changed to "<<hparam.h<<endl;
	}

	hparam.fov = 40.0;
    hparam.aspectratio = hparam.w/hparam.h;
	hparam.angle = tanf(0.5*hparam.fov*M_PI/180.0);
	
	
	Triangle htriangles[MAX_TRIANGLE];
	Light hlights[MAX_LIGHT];
	parse("prism_oct_no_light.asc", htriangles, hparam.n_triangles);
	
	hparam.n_lights = 1;
	
	hlights[0].pos = Vec(1.6,2.749,10.75);
	hlights[0].color = Vec(1,1,1);
	hlights[0].x_vec = Vec(1.2,0,0);
	hlights[0].y_vec = Vec(0,0,1.2);
	hlights[0].n_x = 9;
	hlights[0].n_y = 9;
	
	gpuErrchk(hipSetDevice(0));	

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ctriangles),  htriangles,   sizeof(Triangle)*MAX_TRIANGLE));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cparam) , &hparam, sizeof(Parameter)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(clights), &hlights, sizeof(Light)*MAX_LIGHT));
	
	Vec *c;
	Vec *d_c;

	c = (Vec*)malloc((hparam.w)*(hparam.h)*sizeof(Vec));

	gpuErrchk(hipMalloc((void**) &d_c, (hparam.w)*(hparam.h)*sizeof(Vec)));

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(hparam.w/BLOCK_SIZE,hparam.h/BLOCK_SIZE);

	
	hiprandState_t* states;
	gpuErrchk(hipMalloc((void**) &states, (hparam.w)*(hparam.h) * sizeof(hiprandState_t)));
	init_rand<<<dimGrid,dimBlock>>>(states, time(0));
	gpuErrchk( hipPeekAtLastError());
	

	path_tracing<<<dimGrid,dimBlock>>>(d_c);
	gpuErrchk( hipPeekAtLastError());
	
	
	gpuErrchk(hipMemcpy(c, d_c, (hparam.w)*(hparam.h)*sizeof(Vec), hipMemcpyDeviceToHost));
	
	FILE *f = fopen("image_ray.ppm", "w");         // Write image to PPM file.
   	fprintf(f, "P3\n%d %d\n%d\n", hparam.w, hparam.h, 255);
   	for (int i=0; i<(hparam.w)*(hparam.h); i++) {
		fprintf(f,"%d %d %d ", toInt(c[i].x), toInt(c[i].y), toInt(c[i].z));
	}
	fclose(f);
	
	free(c);
	hipFree(d_c);
	hipFree(states);
	
	
	
	return 0;
}
